#include "hip/hip_runtime.h"
/// \file CudaManager.cu
/// \author Johannes de Fine Licht (johannes.definelicht@cern.ch)

#include "VecGeom/management/CudaManager.h"

#include <stdio.h>

#include "VecGeom/backend/cuda/Backend.h"

namespace vecgeom {
inline namespace cuda {

__global__ void InitDeviceCompactPlacedVolBufferPtrCudaKernel(void *gpu_ptr)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::gCompactPlacedVolBuffer = (vecgeom::cuda::VPlacedVolume *)gpu_ptr;
}

void InitDeviceCompactPlacedVolBufferPtr(void *gpu_ptr)
{
  InitDeviceCompactPlacedVolBufferPtrCudaKernel<<<1, 1>>>(gpu_ptr);
  vecgeom::cxx::CudaAssertError();
}

__global__ void InitDeviceNavIndexPtrCudaKernel(void *gpu_ptr, int maxdepth)
{
  // gpu_ptr is some pointer on the device that was allocated by some other means
  globaldevicegeomdata::gNavIndex = (NavIndex_t *)gpu_ptr;
  globaldevicegeomdata::gMaxDepth = maxdepth;
}

void InitDeviceNavIndexPtr(void *gpu_ptr, int maxdepth)
{
  InitDeviceNavIndexPtrCudaKernel<<<1, 1>>>(gpu_ptr, maxdepth);
}

__global__ void CudaManagerPrintGeometryKernel(vecgeom::cuda::VPlacedVolume const *const world)
{
  printf("Geometry loaded on GPU:\n");
  world->PrintContent();
}

void CudaManagerPrintGeometry(vecgeom::cuda::VPlacedVolume const *const world)
{
  CudaManagerPrintGeometryKernel<<<1, 1>>>(world);
  cxx::CudaAssertError();
  hipDeviceSynchronize();
}
}
} // End namespace vecgeom
